#include <iostream>
#include <cstdio>
#include <complex>
//#include <thrust/complex.h>
#include <hip/hip_complex.h>
using namespace std;
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <math.h>


#define M_PI 3.14159265358979323846 // Pi constant with double precision

__host__ __device__ static __inline__ hipDoubleComplex cuCexp(hipDoubleComplex x)
{
	double factor = exp(hipCreal(x));
	return make_hipDoubleComplex(factor * cos(hipCimag(x)), factor * sin(hipCimag(x)));
}

/*
Separate even/odd elements to lower/upper halves of array respectively.
Defined as __device__ function as it will be acessed by the fft kernel
*/

__device__ void separate (hipDoubleComplex* a, int n, hipDoubleComplex* b)
{
   
  //cudaMalloc((void **)&b,(n*sizeof(cuDoubleComplex))/2);  // get temp heap storage
  for(int i=0; i<n/2; i++)    // copy all odd elements to heap storage
      b[i] = a[i*2 + 1];
  for(int i=0; i<n/2; i++)    // copy all even elements to lower-half of a[]
      a[i] = a[i*2];
  for(int i=0; i<n/2; i++)    // copy all odd (from heap) to upper-half of a[]
      a[i+n/2] = b[i];
                  // delete heap storage
}

__global__ void fft (hipDoubleComplex* X_d, int local_n, int numBlock, int block_size, hipDoubleComplex* b)
{
    
  if(local_n < 2) {
      // bottom of recursion.
      // Do nothing here, because already X[0] = x[0]
  } else {
      separate(X_d,local_n,b);      // all evens to lower half, all odds to upper half
      __syncthreads();
      fft <<< 1, 1 >>> (X_d, local_n/2, 1, 1, b);   // recurse even items
      __syncthreads();
      fft <<< 1, 1 >>> (X_d+local_n/2, local_n/2, 1, 1, b);   // recurse odd  items
      __syncthreads();
      // combine results of two half recursions
      for(int k=0; k<local_n/2; k++) {
          hipDoubleComplex e = X_d[k];   // even
          hipDoubleComplex o = X_d[k + local_n/2];   // odd
                       // w is the "twiddle-factor"
          __syncthreads();
          hipDoubleComplex w = cuCexp( make_hipDoubleComplex(0,-2.*M_PI*(double)(k)/((double)(local_n))) );
          X_d[k] = hipCadd(e, hipCmul(w, o));
          __syncthreads();
          X_d[k + local_n/2] = hipCsub(e, hipCmul(w, o));
          __syncthreads();
      }
  }
  //cudaFre
}

// kernel for generating large sized sampling data using the GPU
__global__ void datagen (hipDoubleComplex* x_d, int n_local, int nSamples)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  const int nFreqs = 5;
  double freq[nFreqs] = { 2, 5, 11, 17, 29 }; // known freqs for testing

  // generate samples for testing
  for(int i=0; i<n_local; i++) {
      x_d[idx+i] = make_hipDoubleComplex(0,0);
      // sum several known sinusoids into x[]
      for(int j=0; j<nFreqs; j++)
          x_d[i+idx] = hipCadd(x_d[i+idx], make_hipDoubleComplex(sin( 2*M_PI*(float)(freq[j])*(float)(i+idx)/((float)nSamples) ), 0));
  }
}

int main() {

  /*
  NOTE:
  Keep nSamples, block_size and n_local a power of 2
  otherwise bad things will happen
  */

  const int nSamples = 512;        // total number of sampling points
  int n_local = 1;                   // local number of samples in each thread
                                     // constraint to keep it a power of 2
  double nSeconds = 1.0;                         // total time for sampling
  double sampleRate = nSamples / nSeconds;       // n Hz = n / second
  double freqResolution = sampleRate / nSamples; // freq step in FFT result

  hipError_t errorcode = hipSuccess;          // for cuda error mgmt

  complex<double> x_h[nSamples];                // storage for sample data(host copy)
  complex<double> X_h[nSamples];                // storage for FFT answer(host copy)

  hipDoubleComplex* x_d;                          // storage for sample data(device copy)
  // Assign memory for device copy of sample data storage with error handling
  if (( errorcode = hipMalloc((void **)&x_d,nSamples*sizeof(hipDoubleComplex)))!= hipSuccess)
  {
    cout << "cudaMalloc(): " << hipGetErrorString(errorcode) << endl;
    exit(1);
  }

  hipDoubleComplex* X_d;                          // storage for FFT answer(device copy)
  // Assign memory for device copy of FFT answer storage with error handling
  if (( errorcode = hipMalloc((void **)&X_d,nSamples*sizeof(hipDoubleComplex)))!= hipSuccess)
  {
    cout << "cudaMalloc(): " << hipGetErrorString(errorcode) << endl;
    exit(1);
  }

  int block_size = 256;
  int numBlock = nSamples/(n_local*block_size);  // Essential that everything is a power of 2

/*****************************************************************************************/

  datagen <<< numBlock, block_size >>> (x_d, n_local, nSamples); // Kernel call for data generation

/*****************************************************************************************/

  // Copy the device copy of generated data to host copy with error handling
  if((errorcode = hipMemcpy(x_h, x_d, nSamples*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost))!=hipSuccess)
  {
    cout << "3cudaMemcpy(): " << hipGetErrorString(errorcode) << endl;
    exit(1);
  }

  for(int i=0; i<nSamples; i++)  X_h[i] = x_h[i];        // copy into X[] for FFT work & result

  hipFree(x_d);          // Needed as fresh GPU computations will begin post this point
  


  hipDoubleComplex* b;
  hipMalloc((void **)&b,(1024*sizeof(hipDoubleComplex))/2);
  // thrust::complex<double>* ex;
  // ex = thrust::complex<double>(1.0,1.0);
  for(int i=0; i<nSamples; i++) cout<< (x_h[i])<<"\n";	
  // Copy the host copy of original FFT answer storage(X_h) to device copy(X_d)
  if((errorcode = hipMemcpy(X_d, X_h, nSamples*sizeof(hipDoubleComplex), hipMemcpyHostToDevice))!=hipSuccess)
  {
    cout << "2cudaMemcpy(): " << hipGetErrorString(errorcode) << endl;
    exit(1);
  }

/****************************************************************************************/

  fft <<< 1, 1 >>> (X_d, nSamples, numBlock, block_size, b); // Kernel call for fft computation

/****************************************************************************************/

  if((errorcode = hipMemcpy(X_h, X_d, nSamples*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost))!=hipSuccess)
  {
    cout << "1cudaMemcpy(): " << hipGetErrorString(errorcode) << endl;
    exit(1);
  }

  
  ::freopen("FFT_out.txt", "w", stdout);

  cout<<"  n\tx[]\tX[]\tf\n";       // header line
  // loop to print values
  for(int i=0; i<nSamples; i++)
  {
      cout<<i<<"\t";
      cout<< ((x_h[i]).real()) << "\t";
      cout<< abs(X_h[i]) << "\t" ;
      cout<<i*freqResolution<<"\n";
  }

  ::fclose(stdout);
 
  hipFree(X_d); 
  hipFree(b);  
return 0;
}
